#include "hip/hip_runtime.h"
 
#include "hiprand/hiprand_kernel.h" 

#include <stdio.h>
#include <time.h>


__global__ void kernel_set_random(hiprandState *curand_states,int width,int height,long clock_for_rand)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;

    if(x<0 || x>width)
    {
        return;
    }
    hiprand_init(clock_for_rand,x,0,&curand_states[x]);
}

__global__ void kernel_random(float *dev_random_array,int width,int height,hiprandState *curand_states)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;

    if(x<0 || x>width)
    {
        return;
    }

    for(int y=0;y<height;y++)
    {
        int pos = y*width + x;
        dev_random_array[pos] = abs(hiprand_uniform(curand_states+x));
    }
}


int main()
{
    const int array_size_width = 10;
    const int array_size_height = 10;
    float random_array[array_size_width*array_size_height];
    for(int i=0;i<array_size_width*array_size_height;i++)
    {
        random_array[i] = 0;
    }

    //error status
    hipError_t cuda_status;

    //only chose one GPU
    cuda_status = hipSetDevice(0);
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"hipSetDevice failed! Do you have a CUDA-Capable GPU installed?");
        return 0;
    }

    float *dev_random_array;
    hiprandState *dev_states;

     //allocate memory on the GPU
    cuda_status = hipMalloc((void**)&dev_random_array,sizeof(float)*array_size_width*array_size_height);
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"dev_random_array hipMalloc Failed");
        exit( EXIT_FAILURE );
    }
    cuda_status = hipMalloc((void **)&dev_states,sizeof(hiprandState)*array_size_width*array_size_height);
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"dev_states hipMalloc Failed");
        exit( EXIT_FAILURE );
    }

    long clock_for_rand = clock();


    

    dim3 threads(16,1);
    dim3 grid((array_size_width+threads.x-1)/threads.x,1);  

    kernel_set_random<<<grid,threads>>>(dev_states,array_size_width,array_size_height,clock_for_rand);

    printf("The first time \n");
    {
        kernel_random<<<grid,threads>>>(dev_random_array,array_size_width,array_size_height,dev_states);

        //copy out the result
        cuda_status = hipMemcpy(random_array,dev_random_array,sizeof(float)*array_size_width*array_size_height,hipMemcpyDeviceToHost);//dev_depthMap
        if(cuda_status != hipSuccess)
        {
            fprintf(stderr,"hipMemcpy Failed");
            exit( EXIT_FAILURE );
        }

        for(int i=0;i<array_size_width*array_size_height;i++)
        {
            printf("%f\n",random_array[i]);
        }
    }
    printf("------------------------------------------------------- \n");
    printf("The second time \n");
    {
        kernel_random<<<grid,threads>>>(dev_random_array,array_size_width,array_size_height,dev_states);

        //copy out the result
        cuda_status = hipMemcpy(random_array,dev_random_array,sizeof(float)*array_size_width*array_size_height,hipMemcpyDeviceToHost);//dev_depthMap
        if(cuda_status != hipSuccess)
        {
            fprintf(stderr,"hipMemcpy Failed");
            exit( EXIT_FAILURE );
        }

        for(int i=0;i<array_size_width*array_size_height;i++)
        {
            printf("%f\n",random_array[i]);
        }
    }

    //free
    hipFree(dev_random_array);
    hipFree(dev_states);
    return 0;
}