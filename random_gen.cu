#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" 
#include "hiprand/hiprand_kernel.h" 

#include <stdio.h>
#include <time.h>


__global__ void kernel_set_random(hiprandState *curand_states,int size,long clock_for_rand)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    
    if(x<0 || x>size)
    {
        return;
    }

    hiprand_init(clock_for_rand,x,0,&curand_states[x]);
}

__global__ void kernel_random(float *dev_random_array,int height,int width,hiprandState *curand_states)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(x<0 || x>width || y<0 || y>height)
    {
        return;
    }

    int pos = y*width + x;

    dev_random_array[pos] = abs(hiprand_uniform(&curand_states[x]));
}


int main()
{
    const int array_size_width = 1000;
    const int array_size_height = 1000;
    float random_array[array_size_width*array_size_height];
    for(int i=0;i<array_size_width*array_size_height;i++)
    {
        random_array[i] = 0;
    }

    //error status
    hipError_t cuda_status;

    //only chose one GPU
    cuda_status = hipSetDevice(0);
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"hipSetDevice failed! Do you have a CUDA-Capable GPU installed?");
        return 0;
    }

    float *dev_random_array;
    hiprandState *dev_states;

     //allocate memory on the GPU
    cuda_status = hipMalloc((void**)&dev_random_array,sizeof(float)*array_size_width*array_size_height);
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"dev_reference_image hipMalloc Failed");
        exit( EXIT_FAILURE );
    }
    cuda_status = hipMalloc((void **)&dev_states,sizeof(hiprandState)*array_size_width);
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"dev_states hipMalloc Failed");
        exit( EXIT_FAILURE );
    }

    long clock_for_rand = clock();

    dim3 threads_x(16,1);
    dim3 grid_x(max((array_size_width+threads_x.x-1)/threads_x.x,1),1);
    kernel_set_random<<<grid_x,threads_x>>>(dev_states,array_size_width,clock_for_rand);

    dim3 threads(16,16);
    dim3 grid(max((array_size_width+threads.x-1)/threads.x,1),max((array_size_height+threads.y-1)/threads.y,1));  
    kernel_random<<<grid,threads>>>(dev_random_array,array_size_width,array_size_height,dev_states);

    //copy out the result
    cuda_status = hipMemcpy(random_array,dev_random_array,sizeof(float)*array_size_width*array_size_height,hipMemcpyDeviceToHost);//dev_depthMap
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"hipMemcpy Failed");
        exit( EXIT_FAILURE );
    }

    for(int i=0;i<array_size_width*array_size_height;i++)
    {
        printf("%f\n",random_array[i]);
    }

    //free
    hipFree(dev_random_array);
    hipFree(dev_states);
    return 0;
}